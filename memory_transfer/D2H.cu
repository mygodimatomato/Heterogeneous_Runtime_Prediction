#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <ctime>
#include <vector>
#include <iomanip>  // for std::setw

#define MATRIX_SIZE 100
#define MAX_MATRICES 10000
#define device_id 3  // Change this to your desired GPU device ID

int main() {
    hipSetDevice(device_id);
    srand(time(0));
    std::cout << std::fixed << std::setprecision(6);
    std::cout << "Num_Matrices\tDtoH_Time_ms" << std::endl;

    for (int num_matrices = 1; num_matrices <= MAX_MATRICES; num_matrices += 10) {
        size_t total_elements = num_matrices * MATRIX_SIZE * MATRIX_SIZE;
        size_t bytes = total_elements * sizeof(float);

        // Allocate and fill host memory
        float* h_data = new float[total_elements];
        float* h_result = new float[total_elements];  // for device-to-host transfer
        for (size_t i = 0; i < total_elements; ++i) {
            h_data[i] = static_cast<float>(rand()) / RAND_MAX;
        }

        // Allocate device memory
        float* d_data;
        hipError_t err = hipMalloc(&d_data, bytes);
        if (err != hipSuccess) {
            std::cerr << "hipMalloc failed at n=" << num_matrices << ": "
                      << hipGetErrorString(err) << std::endl;
            delete[] h_data;
            delete[] h_result;
            break;
        }

        // CUDA events for Host-to-Device
        // cudaEvent_t h2d_start, h2d_stop;
        // cudaEventCreate(&h2d_start);
        // cudaEventCreate(&h2d_stop);
        // cudaEventRecord(h2d_start);
        hipMemcpy(d_data, h_data, bytes, hipMemcpyHostToDevice);
        // cudaEventRecord(h2d_stop);
        // cudaEventSynchronize(h2d_stop);

        // float h2d_elapsed = 0.0f;
        // cudaEventElapsedTime(&h2d_elapsed, h2d_start, h2d_stop);

        // CUDA events for Device-to-Host
        hipEvent_t d2h_start, d2h_stop;
        hipEventCreate(&d2h_start);
        hipEventCreate(&d2h_stop);
        hipEventRecord(d2h_start);
        hipMemcpy(h_result, d_data, bytes, hipMemcpyDeviceToHost);
        hipEventRecord(d2h_stop);
        hipEventSynchronize(d2h_stop);

        float d2h_elapsed = 0.0f;
        hipEventElapsedTime(&d2h_elapsed, d2h_start, d2h_stop);

        // Output
        std::cout << std::setw(12) << num_matrices << "\t"
                  << d2h_elapsed << std::endl;

        // Cleanup
        hipEventDestroy(d2h_start);
        hipEventDestroy(d2h_stop);
        hipFree(d_data);
        delete[] h_data;
        delete[] h_result;
    }

    return 0;
}
