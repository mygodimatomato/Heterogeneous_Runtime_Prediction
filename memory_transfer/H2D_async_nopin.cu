#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <iomanip>

#define MATRIX_SIZE   10
#define MAX_MATRICES  20000
#define GPU_ID        0          // pick your device

int main() {
    hipSetDevice(GPU_ID);

    /* ---------- 1.  Create one stream and one reusable pinned buffer ---------- */
    hipStream_t stream;
    hipStreamCreate(&stream);

    const size_t max_elems  = MAX_MATRICES * MATRIX_SIZE * MATRIX_SIZE;
    const size_t max_bytes  = max_elems * sizeof(float);

    float* h_data = nullptr;                 // page-locked host memory
    // cudaMallocHost(&h_data, max_bytes);      // ALWAYS pinned if you want overlap
    // call malloc and assign a size = max_bytes of h_data
    h_data = (float*)malloc(max_bytes); // Use malloc for simplicity in this example

    std::cout << std::fixed << std::setprecision(6)
              << "Num_Matrices\tCopy_Time_ms\n";

    /* ---------- 2.  Timing loop ---------- */
    for (int n = 1; n <= MAX_MATRICES; n += 10) {
        size_t elems  = n * MATRIX_SIZE * MATRIX_SIZE;
        size_t bytes  = elems * sizeof(float);

        /* fill the portion we’ll send */
        for (size_t i = 0; i < elems; ++i)
            h_data[i] = static_cast<float>(rand()) / RAND_MAX;

        float* d_data = nullptr;
        hipMalloc(&d_data, bytes);          // sync alloc OK in this micro-bench

        /* events live in the SAME stream so they bracket only the async copy */
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipEventRecord(start, stream);
        hipMemcpyAsync(d_data, h_data, bytes,
                        hipMemcpyHostToDevice, stream);
        hipEventRecord(stop, stream);

        hipStreamSynchronize(stream);       // wait just once per iteration

        float ms = 0.f;
        hipEventElapsedTime(&ms, start, stop);

        std::cout << std::setw(12) << n << '\t' << ms << '\n';

        hipEventDestroy(start);
        hipEventDestroy(stop);
        hipFree(d_data);
    }

    /* ---------- 3.  Cleanup ---------- */
    hipHostFree(h_data);
    hipStreamDestroy(stream);
    return 0;
}
