#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <iomanip>

#define MATRIX_SIZE   10
#define MAX_MATRICES  20000
#define GPU_ID        0

int main() {
    hipSetDevice(GPU_ID);

    /* ---------- 1.  Stream + reusable pinned host buffer ---------- */
    hipStream_t stream;
    hipStreamCreate(&stream);

    const size_t max_elems = MAX_MATRICES * MATRIX_SIZE * MATRIX_SIZE;
    const size_t max_bytes = max_elems * sizeof(float);

    float* h_data = nullptr;                  // page-locked host memory
    hipHostMalloc(&h_data, max_bytes, hipHostMallocDefault);       // so the copy can overlap

    std::cout << std::fixed << std::setprecision(6)
              << "Num_Matrices\tCopy_Time_ms\n";
              
    // run the cudaMemcpyAsync for warmup
    float* warmup = nullptr;
    hipMalloc(&warmup, max_bytes);           // sync alloc OK in this micro-bench
    hipMemcpyAsync(warmup, h_data, max_bytes,
                    hipMemcpyHostToDevice, stream);
    hipStreamSynchronize(stream);
    

    /* ---------- 2.  Timing loop ---------- */
    for (int n = 1; n <= MAX_MATRICES; n += 10) {
        size_t elems = n * MATRIX_SIZE * MATRIX_SIZE;
        size_t bytes = elems * sizeof(float);

        // /* fill the portion we’ll send */
        // for (size_t i = 0; i < elems; ++i)
        //     h_data[i] = static_cast<float>(rand()) / RAND_MAX;

        /* seed device buffer with something to copy back */
        float* d_data = nullptr;
        hipMalloc(&d_data, bytes);
        hipMemcpyAsync(d_data, h_data, bytes,          // cheap H2D warm-up
                        hipMemcpyHostToDevice, stream);
        hipStreamSynchronize(stream);                  // ensure data is ready

        /* events bracket ONLY the async D2H copy */
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipEventRecord(start, stream);
        hipMemcpyAsync(h_data,      // ← destination now HOST
                        d_data,      // ← source is DEVICE
                        bytes,
                        hipMemcpyDeviceToHost,  // ← D2H direction
                        stream);
        hipEventRecord(stop, stream);

        hipStreamSynchronize(stream);          // wait once per iteration

        float ms = 0.f;
        hipEventElapsedTime(&ms, start, stop);

        std::cout << std::setw(12) << n << '\t' << ms << '\n';

        hipEventDestroy(start); hipEventDestroy(stop);
        hipFree(d_data);
    }

    /* ---------- 3.  Cleanup ---------- */
    hipHostFree(h_data);
    hipStreamDestroy(stream);
    return 0;
}
