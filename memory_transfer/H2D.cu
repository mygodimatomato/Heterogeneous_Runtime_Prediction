#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <ctime>
#include <vector>
#include <iomanip>  // for std::setw

#define MATRIX_SIZE 10
#define MAX_MATRICES 20000
#define GPU_ID 0  // Change this to your desired GPU device ID

int main() {
    hipSetDevice(GPU_ID);

    const size_t max_elems  = MAX_MATRICES * MATRIX_SIZE * MATRIX_SIZE;
    const size_t max_bytes  = max_elems * sizeof(float);

    float* h_data = nullptr;  // Host memory
    h_data = (float*)malloc(max_bytes);

    std::cout << std::fixed << std::setprecision(6)
              << "Num_Matrices\tCopy_Time_ms\n";

    for (int n = 1; n <= MAX_MATRICES; n += 10) {
        size_t elems = n * MATRIX_SIZE * MATRIX_SIZE;
        size_t bytes = elems * sizeof(float);

        // Allocate and fill host memory
        for (size_t i = 0; i < elems; ++i) {
            h_data[i] = static_cast<float>(rand()) / RAND_MAX;
        }

        // Allocate device memory
        float* d_data = nullptr;
        hipMalloc(&d_data, bytes);

        // Setup CUDA events
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        // Record timing for cudaMemcpy
        hipEventRecord(start);
        hipMemcpy(d_data, h_data, bytes, hipMemcpyHostToDevice);
        hipEventRecord(stop);
        hipEventSynchronize(stop);

        float ms = 0;
        hipEventElapsedTime(&ms, start, stop);

        // Print result
        std::cout << std::setw(12) << n << "\t" << ms << std::endl;

        // Cleanup
        hipEventDestroy(start);
        hipEventDestroy(stop);
        hipFree(d_data);
    }

    free(h_data);
    return 0;
}
